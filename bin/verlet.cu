#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <cutil_math.h>

#include <cassert>

float4* X[2];
float4* X_last[2];

float4 * X_in, *X_out;
float4 * X_last_in, *X_last_out;

extern int readID, writeID;
__global__ void verlet(	float4 * pos_vbo, float4 * g_pos_in, float4 * g_pos_old_in, float4 * g_pos_out, float4 * g_pos_old_out, 
							int2 texsize, float2 step,  float damp, float mass, float dt, float2 inv_cloth_size);

void InitCUDA(const unsigned int size) {
	const unsigned int num_threads = size;
	const unsigned int mem_size = sizeof(float4) * num_threads;
	
	// allocate device memory for float4 version
	cutilSafeCall(hipMalloc((void**) &X[0], mem_size));	// positions
	cutilSafeCall(hipMalloc((void**) &X[1], mem_size));	// positions
	cutilSafeCall(hipMalloc((void**) &X_last[0], mem_size));	// old positions
	cutilSafeCall(hipMalloc((void**) &X_last[1], mem_size));	// old positions		
}

void ShutdownCUDA()
{
	// cleanup memory
	if (X[0] != NULL) 
	{
		cutilSafeCall(hipFree(X[0]));
		cutilSafeCall(hipFree(X[1]));
		X[0] = NULL;
		X[1] = NULL;
	}

	if (X_last[0] != NULL)
	{
		cutilSafeCall(hipFree(X_last[0]));
		cutilSafeCall(hipFree(X_last[1]));
		X_last[0] = NULL;
		X_last[1] = NULL;
	}
}

void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n); 
    numBlocks = (n % numThreads != 0) ? (n / numThreads + 1) : (n / numThreads);
}  

void UploadCUDA(float * positions, float * positions_old, const int size)
{
	static bool start = true;

	assert(X[0] != NULL); 
	assert(X_last[0] != NULL); 

	const unsigned int num_threads = size;
	const unsigned int mem_size = sizeof(float4) * num_threads;

	X_in  = X[readID];	
	X_out = X[writeID];
	X_last_in  = X_last[readID];	
	X_last_out = X_last[writeID];
	
	if (start)
	{
		cutilSafeCall(hipMemcpy(X_in, positions,  mem_size, hipMemcpyHostToDevice));
		cutilSafeCall(hipMemcpy(X_last_in, positions_old, mem_size, hipMemcpyHostToDevice));
		cutilCheckMsg("Cuda memory copy host to device failed.");
		start=false;
	} 

	int tmp=readID;
	readID = writeID;
	writeID=tmp;
}

void VerletCUDA(float4 * pos_vbo, int2 texsize, float2 step, const float & damp, const float & mass, float dt, float2 inv_cloth_size)
{   
	// setup execution parameters 
	uint numThreads, numBlocks;
	uint numParticles = texsize.x*texsize.y;

	computeGridSize(numParticles, 256, numBlocks, numThreads);

//	printf("%3d particles, %3d blocks, %3d threads\n", numParticles, numBlocks, numThreads);

	// execute the kernel
	//	printf("numParticles: %d,   numThreads: %d   numBlocks: %d\n", numParticles, numThreads, numBlocks);
	verlet<<< numBlocks, numThreads >>>(pos_vbo, X_in, X_last_in, X_out, X_last_out, texsize, step, damp, mass, dt, inv_cloth_size);

	// stop the CPU until the kernel has been executed
	hipDeviceSynchronize();

	// check if kernel execution generated and error
	cutilCheckMsg("Cuda kernel execution failed.");
}
