#include "hip/hip_runtime.h"
#include <cutil_math.h>
#include <hip/hip_math_constants.h>

//#define USE_SMEM
#define BLOCKSIZE (128 + 2) * (128 + 2)

 
const __device__ float	KsStruct = 50.75f,KdStruct = -0.25f, 
	KsShear = 50.75f,KdShear = -0.25f,
	KsBend = 50.95f,KdBend = -0.25f;



__device__ int2 getNextNeighbor(int n, float& ks, float& kd) { 
    //structural springs (adjacent neighbors)
    //        o
    //        |
    //     o--m--o
    //        |
    //        o
    if(n<4) {
       ks = KsStruct;
       kd = KdStruct;
    }
	if (n == 0)	return make_int2( 1,  0);
	if (n == 1)	return make_int2( 0, -1);
	if (n == 2)	return make_int2(-1,  0);
	if (n == 3)	return make_int2( 0,  1);
	
	//shear springs (diagonal neighbors)
	//     o  o  o
	//      \   /
	//     o  m  o
	//      /   \
	//     o  o  o
	if(n<8) {
       ks = KsShear;
       kd = KdShear;
    }
	if (n == 4) return make_int2( 1,  -1);
	if (n == 5) return make_int2( -1, -1);	
	if (n == 6) return make_int2(-1,  1);
	if (n == 7) return make_int2( 1,  1);
	
	//bend spring (adjacent neighbors 1 node away)
	//
	//o   o   o   o   o
	//        | 
	//o   o   |   o   o
	//        |   
	//o-------m-------o
	//        |  
	//o   o   |   o   o
	//        |
	//o   o   o   o   o 
	if(n<12) {
       ks = KsBend;
       kd = KdBend;
    }
	if (n == 8)	return make_int2( 2, 0);
	if (n == 9) return make_int2( 0, -2);
	if (n ==10) return make_int2(-2, 0);
	if (n ==11) return make_int2( 0, 2);
}

///////////////////////////////////////////////////////////////////////////////
//! kernel for cloth simulating via verlet integration
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void verlet(	float4 * pos_vbo, float4 * g_pos_in, float4 * g_pos_old_in, float4 * g_pos_out, float4 * g_pos_old_out, 
							int2 texsize, float2 step,  float damp, float mass, float dt, float2 inv_cloth_size)
{
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = index % texsize.x; 
	int iy = index / texsize.x; 

	//g_pos[index] = make_float4(threadIdx.x, blockIdx.x, blockDim.x, index);
	//return;
    float ks=0, kd=0;
#ifdef USE_SMEM
    __shared__ float4 smem_pos[BLOCKSIZE];
    __shared__ float4 smem_last_pos[BLOCKSIZE];

	int ix_smem = threadIdx.x % texsize.x;  
	int iy_smem = threadIdx.x / texsize.x; 

	smem_pos[threadIdx.x] = g_pos_in[index]; 
	smem_last_pos[threadIdx.x] = g_pos_old_in[index]; 

	for (int k = 0; k < 12; k++)
	{
		int2 coord = getNextNeighbor(k, ks, kd);
		int j = coord.x;
		int i = coord.y;

		if (((iy_smem + i) < 0) || ((iy_smem + i) > (texsize.x - 1)))
			continue;

		if (((ix_smem + j) < 0) || ((ix_smem + j) > (texsize.x - 1)))
			continue;

		int index_neigh_smem = (iy_smem + i) * texsize.x + ix_smem + j;
		int index_neigh = (iy + i) * texsize.x + ix + j;

		smem_pos[index_neigh_smem] = g_pos_in[index_neigh]; 
		smem_last_pos[index_neigh_smem] = g_pos_old_in[index_neigh]; 
	}

	__syncthreads();

	volatile float4 posData = smem_pos[threadIdx.x];    // ensure coalesced read
    volatile float4 posOldData = smem_last_pos[threadIdx.x];
#else
	volatile float4 posData = g_pos_in[index];    // ensure coalesced read
    volatile float4 posOldData = g_pos_old_in[index];
#endif


    float3 pos = make_float3(posData.x, posData.y, posData.z);
    float3 pos_old = make_float3(posOldData.x, posOldData.y, posOldData.z);
	float3 vel = (pos - pos_old) / dt;
	 
	const float3 gravity=make_float3(0.0f,-0.00981f,0.0f); 
	float3 force = gravity*mass + vel*damp;
	  
	if (index==0 || index== (texsize.x - 1.0))
		 mass = 0.0;

	
	for (int k = 0; k < 12; k++)
	{
		int2 coord = getNextNeighbor(k, ks, kd);//nextNeigh(k);
		int j = coord.x;
		int i = coord.y;

#ifdef USE_SMEM
		if (((iy_smem + i) < 0) || ((iy_smem + i) > (texsize.x - 1)))
			continue;

		if (((ix_smem + j) < 0) || ((ix_smem + j) > (texsize.x - 1)))
			continue;

		int index_neigh_smem = (iy_smem + i) * texsize.x + ix_smem + j;

		volatile float4 pos_neighData = smem_pos[index_neigh_smem];
		volatile float4 pos_lastData = smem_last_pos[index_neigh_smem];
		
#else
		if (((iy + i) < 0) || ((iy + i) > (texsize.y - 1)))
			continue;

		if (((ix + j) < 0) || ((ix + j) > (texsize.x - 1)))
			continue;

		int index_neigh = (iy + i) * texsize.x + ix + j;

		volatile float4 pos_neighData = g_pos_in[index_neigh];
		volatile float4 pos_lastData = g_pos_old_in[index_neigh];
#endif
		float3 p2 = make_float3(pos_neighData.x, pos_neighData.y, pos_neighData.z);
        float3 p2_last = make_float3(pos_lastData.x, pos_lastData.y, pos_lastData.z);
		float2 coord_neigh = make_float2(ix + j, iy + i) * step;
		
		float rest_length = length(make_float2(coord.x*inv_cloth_size.x, coord.y*inv_cloth_size.y));
		 
		 
		float3 v2 = (p2- p2_last)/dt;
		float3 deltaP = pos - p2;	
		float3 deltaV = vel - v2;	 
		float dist = length(deltaP);
				
		float   leftTerm = -ks * (dist-rest_length);
		float  rightTerm = kd * (dot(deltaV, deltaP)/dist);		
		float3 springForce = (leftTerm + rightTerm)*normalize(deltaP);
		force += springForce;	
	}

	float3 acc = make_float3(0, 0, 0);
	if(mass!=0)
		acc = force / mass;

	
	if(pos.y<0)
	   pos.y=0;

	// verlet
	float3 tmp = pos; 
	pos = pos * 2 - pos_old + acc * dt * dt;
	pos_old = tmp;

	syncthreads();

	pos_vbo[index] = make_float4(pos.x, pos.y, pos.z, posData.w);
	g_pos_out[index] = make_float4(pos.x, pos.y, pos.z, posData.w);
	g_pos_old_out[index] = make_float4(pos_old.x, pos_old.y, pos_old.z, posOldData.w);

}

